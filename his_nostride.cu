#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define SIZE 1000000
#define RANGE 10

// Time function
double get_clock() {
    struct timeval tv;
    int ok;
    ok = gettimeofday(&tv, NULL);
    if (ok < 0) {
        printf("gettimeofday error\n");
        exit(1);
    }
    return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

// GPU kernel to compute histogram
__global__ void computeHistogramGPU(int *data, int *histogram, int size, int range) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < size) {
        // Atomic add to ensure thread-safe updates to the histogram
        atomicAdd(&histogram[data[idx]], 1);
    }
}

// GPU kernel to initialize the histogram to zero
__global__ void initHistogram(int *histogram, int range) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < range) {
        histogram[idx] = 0;  // Set each element to zero
    }
}

int main() {
    int *data = (int *)malloc(sizeof(int) * SIZE);
    int *histogram = (int *)malloc(sizeof(int) * RANGE);
    int *d_data, *d_histogram;

    // Generate random data
    printf("Generated Data:\n");
    for (int i = 0; i < SIZE; i++) {
        data[i] = rand() % RANGE;
        printf("%d ", data[i]);
    }
    printf("\n");

    // Allocate memory on the device
    hipMalloc((void **)&d_data, sizeof(int) * SIZE);
    hipMalloc((void **)&d_histogram, sizeof(int) * RANGE);

    // Copy data to the device
    hipMemcpy(d_data, data, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    // Initialize the histogram on the device to zero
    int blockSize = 256; // Number of threads per block
    int numBlocks = (RANGE + blockSize - 1) / blockSize; // Compute the number of blocks for initialization
    initHistogram<<<numBlocks, blockSize>>>(d_histogram, RANGE);
    
    // Synchronize to ensure initialization is complete
    hipDeviceSynchronize();

    // Measure time
    double t0 = get_clock();

    // Launch the kernel to compute the histogram
    numBlocks = (SIZE + blockSize - 1) / blockSize; // Compute the number of blocks for the main kernel
    computeHistogramGPU<<<numBlocks, blockSize>>>(d_data, d_histogram, SIZE, RANGE);

    // Synchronize to ensure kernel execution is done
    hipDeviceSynchronize();

    double t1 = get_clock();

    // Copy the histogram back to the host
    hipMemcpy(histogram, d_histogram, sizeof(int) * RANGE, hipMemcpyDeviceToHost);

    // Print the histogram
    printf("Histogram:\n");
    for (int i = 0; i < RANGE; i++) {
        printf("%d: %d\n", i, histogram[i]);
    }

    // Print time taken
    printf("Time: %f ns\n", 1000000000.0 * (t1 - t0));

    // Free memory
    free(data);
    free(histogram);
    hipFree(d_data);
    hipFree(d_histogram);

    return 0;
}
